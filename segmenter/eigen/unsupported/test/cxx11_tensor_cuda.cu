#include "hip/hip_runtime.h"
// This file is part of Eigen, a lightweight C++ template library
// for linear algebra.
//
// Copyright (C) 2014 Benoit Steiner <benoit.steiner.goog@gmail.com>
//
// This Source Code Form is subject to the terms of the Mozilla
// Public License v. 2.0. If a copy of the MPL was not distributed
// with this file, You can obtain one at http://mozilla.org/MPL/2.0/.

#define EIGEN_TEST_NO_LONGDOUBLE
#define EIGEN_TEST_NO_COMPLEX
#define EIGEN_TEST_FUNC cxx11_tensor_cuda
#define EIGEN_DEFAULT_DENSE_INDEX_TYPE int
#define EIGEN_USE_GPU


#include "main.h"
#include <unsupported/Eigen/CXX11/Tensor>

using Eigen::Tensor;

void test_cuda_elementwise_small() {
  Tensor<float, 1> in1(Eigen::array<int, 1>(2));
  Tensor<float, 1> in2(Eigen::array<int, 1>(2));
  Tensor<float, 1> out(Eigen::array<int, 1>(2));
  in1.setRandom();
  in2.setRandom();

  std::size_t in1_bytes = in1.size() * sizeof(float);
  std::size_t in2_bytes = in2.size() * sizeof(float);
  std::size_t out_bytes = out.size() * sizeof(float);

  float* d_in1;
  float* d_in2;
  float* d_out;
  hipMalloc((void**)(&d_in1), in1_bytes);
  hipMalloc((void**)(&d_in2), in2_bytes);
  hipMalloc((void**)(&d_out), out_bytes);

  hipMemcpy(d_in1, in1.data(), in1_bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_in2, in2.data(), in2_bytes, hipMemcpyHostToDevice);

  Eigen::CudaStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<float, 1>, Eigen::Aligned> gpu_in1(
      d_in1, Eigen::array<int, 1>(2));
  Eigen::TensorMap<Eigen::Tensor<float, 1>, Eigen::Aligned> gpu_in2(
      d_in2, Eigen::array<int, 1>(2));
  Eigen::TensorMap<Eigen::Tensor<float, 1>, Eigen::Aligned> gpu_out(
      d_out, Eigen::array<int, 1>(2));

  gpu_out.device(gpu_device) = gpu_in1 + gpu_in2;

  assert(hipMemcpyAsync(out.data(), d_out, out_bytes, hipMemcpyDeviceToHost,
                         gpu_device.stream()) == hipSuccess);
  assert(hipStreamSynchronize(gpu_device.stream()) == hipSuccess);

  for (int i = 0; i < 2; ++i) {
    VERIFY_IS_APPROX(
        out(Eigen::array<int, 1>(i)),
        in1(Eigen::array<int, 1>(i)) + in2(Eigen::array<int, 1>(i)));
  }

  hipFree(d_in1);
  hipFree(d_in2);
  hipFree(d_out);
}

void test_cuda_elementwise()
{
  Tensor<float, 3> in1(Eigen::array<int, 3>(72,53,97));
  Tensor<float, 3> in2(Eigen::array<int, 3>(72,53,97));
  Tensor<float, 3> in3(Eigen::array<int, 3>(72,53,97));
  Tensor<float, 3> out(Eigen::array<int, 3>(72,53,97));
  in1.setRandom();
  in2.setRandom();
  in3.setRandom();

  std::size_t in1_bytes = in1.size() * sizeof(float);
  std::size_t in2_bytes = in2.size() * sizeof(float);
  std::size_t in3_bytes = in3.size() * sizeof(float);
  std::size_t out_bytes = out.size() * sizeof(float);

  float* d_in1;
  float* d_in2;
  float* d_in3;
  float* d_out;
  hipMalloc((void**)(&d_in1), in1_bytes);
  hipMalloc((void**)(&d_in2), in2_bytes);
  hipMalloc((void**)(&d_in3), in3_bytes);
  hipMalloc((void**)(&d_out), out_bytes);

  hipMemcpy(d_in1, in1.data(), in1_bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_in2, in2.data(), in2_bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_in3, in3.data(), in3_bytes, hipMemcpyHostToDevice);

  Eigen::CudaStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<float, 3> > gpu_in1(d_in1, Eigen::array<int, 3>(72,53,97));
  Eigen::TensorMap<Eigen::Tensor<float, 3> > gpu_in2(d_in2, Eigen::array<int, 3>(72,53,97));
  Eigen::TensorMap<Eigen::Tensor<float, 3> > gpu_in3(d_in3, Eigen::array<int, 3>(72,53,97));
  Eigen::TensorMap<Eigen::Tensor<float, 3> > gpu_out(d_out, Eigen::array<int, 3>(72,53,97));

  gpu_out.device(gpu_device) = gpu_in1 + gpu_in2 * gpu_in3;

  assert(hipMemcpyAsync(out.data(), d_out, out_bytes, hipMemcpyDeviceToHost, gpu_device.stream()) == hipSuccess);
  assert(hipStreamSynchronize(gpu_device.stream()) == hipSuccess);

  for (int i = 0; i < 72; ++i) {
    for (int j = 0; j < 53; ++j) {
      for (int k = 0; k < 97; ++k) {
        VERIFY_IS_APPROX(out(Eigen::array<int, 3>(i,j,k)), in1(Eigen::array<int, 3>(i,j,k)) + in2(Eigen::array<int, 3>(i,j,k)) * in3(Eigen::array<int, 3>(i,j,k)));
      }
    }
  }

  hipFree(d_in1);
  hipFree(d_in2);
  hipFree(d_in3);
  hipFree(d_out);
}

void test_cuda_props() {
  Tensor<float, 1> in1(200);
  Tensor<bool, 1> out(200);
  in1.setRandom();

  std::size_t in1_bytes = in1.size() * sizeof(float);
  std::size_t out_bytes = out.size() * sizeof(bool);

  float* d_in1;
  bool* d_out;
  hipMalloc((void**)(&d_in1), in1_bytes);
  hipMalloc((void**)(&d_out), out_bytes);

  hipMemcpy(d_in1, in1.data(), in1_bytes, hipMemcpyHostToDevice);

  Eigen::CudaStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<float, 1>, Eigen::Aligned> gpu_in1(
      d_in1, 200);
  Eigen::TensorMap<Eigen::Tensor<bool, 1>, Eigen::Aligned> gpu_out(
      d_out, 200);

  gpu_out.device(gpu_device) = (gpu_in1.isnan)();

  assert(hipMemcpyAsync(out.data(), d_out, out_bytes, hipMemcpyDeviceToHost,
                         gpu_device.stream()) == hipSuccess);
  assert(hipStreamSynchronize(gpu_device.stream()) == hipSuccess);

  for (int i = 0; i < 200; ++i) {
    VERIFY_IS_EQUAL(out(i), (std::isnan)(in1(i)));
  }

  hipFree(d_in1);
  hipFree(d_out);
}

void test_cuda_reduction()
{
  Tensor<float, 4> in1(72,53,97,113);
  Tensor<float, 2> out(72,97);
  in1.setRandom();

  std::size_t in1_bytes = in1.size() * sizeof(float);
  std::size_t out_bytes = out.size() * sizeof(float);

  float* d_in1;
  float* d_out;
  hipMalloc((void**)(&d_in1), in1_bytes);
  hipMalloc((void**)(&d_out), out_bytes);

  hipMemcpy(d_in1, in1.data(), in1_bytes, hipMemcpyHostToDevice);

  Eigen::CudaStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<float, 4> > gpu_in1(d_in1, 72,53,97,113);
  Eigen::TensorMap<Eigen::Tensor<float, 2> > gpu_out(d_out, 72,97);

  array<int, 2> reduction_axis;
  reduction_axis[0] = 1;
  reduction_axis[1] = 3;

  gpu_out.device(gpu_device) = gpu_in1.maximum(reduction_axis);

  assert(hipMemcpyAsync(out.data(), d_out, out_bytes, hipMemcpyDeviceToHost, gpu_device.stream()) == hipSuccess);
  assert(hipStreamSynchronize(gpu_device.stream()) == hipSuccess);

  for (int i = 0; i < 72; ++i) {
    for (int j = 0; j < 97; ++j) {
      float expected = 0;
      for (int k = 0; k < 53; ++k) {
        for (int l = 0; l < 113; ++l) {
          expected =
              std::max<float>(expected, in1(i, k, j, l));
        }
      }
      VERIFY_IS_APPROX(out(i,j), expected);
    }
  }

  hipFree(d_in1);
  hipFree(d_out);
}

template<int DataLayout>
void test_cuda_contraction()
{
  // with these dimensions, the output has 300 * 140 elements, which is
  // more than 30 * 1024, which is the number of threads in blocks on
  // a 15 SM GK110 GPU
  Tensor<float, 4, DataLayout> t_left(6, 50, 3, 31);
  Tensor<float, 5, DataLayout> t_right(Eigen::array<int, 5>(3, 31, 7, 20, 1));
  Tensor<float, 5, DataLayout> t_result(Eigen::array<int, 5>(6, 50, 7, 20, 1));

  t_left.setRandom();
  t_right.setRandom();

  std::size_t t_left_bytes = t_left.size()  * sizeof(float);
  std::size_t t_right_bytes = t_right.size() * sizeof(float);
  std::size_t t_result_bytes = t_result.size() * sizeof(float);

  float* d_t_left;
  float* d_t_right;
  float* d_t_result;

  hipMalloc((void**)(&d_t_left), t_left_bytes);
  hipMalloc((void**)(&d_t_right), t_right_bytes);
  hipMalloc((void**)(&d_t_result), t_result_bytes);

  hipMemcpy(d_t_left, t_left.data(), t_left_bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_t_right, t_right.data(), t_right_bytes, hipMemcpyHostToDevice);

  Eigen::CudaStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<float, 4, DataLayout> > gpu_t_left(d_t_left, 6, 50, 3, 31);
  Eigen::TensorMap<Eigen::Tensor<float, 5, DataLayout> > gpu_t_right(d_t_right, 3, 31, 7, 20, 1);
  Eigen::TensorMap<Eigen::Tensor<float, 5, DataLayout> > gpu_t_result(d_t_result, 6, 50, 7, 20, 1);

  typedef Eigen::Map<Eigen::Matrix<float, Dynamic, Dynamic, DataLayout> > MapXf;
  MapXf m_left(t_left.data(), 300, 93);
  MapXf m_right(t_right.data(), 93, 140);
  Eigen::Matrix<float, Dynamic, Dynamic, DataLayout> m_result(300, 140);

  typedef Tensor<float, 1>::DimensionPair DimPair;
  Eigen::array<DimPair, 2> dims;
  dims[0] = DimPair(2, 0);
  dims[1] = DimPair(3, 1);

  m_result = m_left * m_right;
  gpu_t_result.device(gpu_device) = gpu_t_left.contract(gpu_t_right, dims);

  hipMemcpy(t_result.data(), d_t_result, t_result_bytes, hipMemcpyDeviceToHost);

  for (size_t i = 0; i < t_result.dimensions().TotalSize(); i++) {
    if (fabs(t_result.data()[i] - m_result.data()[i]) >= 1e-4) {
      std::cout << "mismatch detected at index " << i << ": " << t_result.data()[i] << " vs " <<  m_result.data()[i] << std::endl;
      assert(false);
    }
  }

  hipFree(d_t_left);
  hipFree(d_t_right);
  hipFree(d_t_result);
}

template<int DataLayout>
void test_cuda_convolution_1d()
{
  Tensor<float, 4, DataLayout> input(74,37